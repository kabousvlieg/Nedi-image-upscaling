#include "hip/hip_runtime.h"
//
// Created by kobus on 2017/04/06.
//

#include "cudaDsp.h"
#include "cpuDsp.h"
#include <opencv2/imgproc.hpp>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <string>
#include <iostream>

#if USE_CUDA
#include <hip/hip_runtime.h>
#include <opencv2/cudacodec.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudabgsegm.hpp>
#include <opencv2/cudalegacy.hpp>
#include <opencv2/cudaobjdetect.hpp>
#include <opencv2/cudafeatures2d.hpp>
#include <opencv2/cudaoptflow.hpp>
#include <opencv2/cudastereo.hpp>
#include <opencv2/cudawarping.hpp>
#endif


#if USE_CUDA

__global__ void convolutionGPU(
		std::vector<cv::cuda::GpuMat> inputChannels,
		std::vector<cv::cuda::GpuMat> outputChannels,
		cv::cuda::GpuMat kernel);

void cudaDsp::executeTiPipeline(cv::Mat& frame, cv::Mat& originalFrame, TiDspParameters params, bool showComparisonWindows)
{
    static bool runOnce = false;
    if (!runOnce)
    {
		int runTimeVersion;
		hipRuntimeGetVersion(&runTimeVersion);
		int driverVersion;
		hipDriverGetVersion(&driverVersion);

		std::cout << "Cuda runtime version : " << runTimeVersion << std::endl;
		std::cout << "Cuda api version : " << driverVersion << std::endl;

		runOnce = true;
    }

    cv::cuda::GpuMat gpuFrame;
    gpuFrame.upload(frame);
    //convertScaleAbs(frame, frame);
    if (showComparisonWindows)
    {
        originalFrame = frame.clone();
    }

    //Image processing pipeline
    if (params.frameAverageOn)
        frameAverage(gpuFrame);
//    if (params.outputSelect == OutputSelect::FrameAverage)
//        gpuFrame.download(outputFrame);

//    if (params.outputSelect == OutputSelect::Nuc)
//        gpuFrame.download(outputFrame);

//    if (params.outputSelect == OutputSelect::TemporalFilter)
//        gpuFrame.download(outputFrame);

//    if (params.outputSelect == OutputSelect::SpatialFilter)
//        gpuFrame.download(outputFrame);

    if (params.edgeEnhanceOn)
    	edgeEnhance(gpuFrame, params);
//    if (params.outputSelect == OutputSelect::EdgeEnhance)
//        gpuFrame.download(outputFrame);

//    if (params.outputSelect == OutputSelect::Stab)
//        gpuFrame.download(outputFrame);

    if (params.eZoomOn)
    {
        eZoom(gpuFrame, params);
        if (showComparisonWindows)
        {
            cpuDsp::eZoom(originalFrame, params);
        }
    }
//    if (params.outputSelect == OutputSelect::Ezoom)
//        frame.copyTo(outputFrame);

    if (params.histogramStretchOn)
        equalizeHistogram(gpuFrame);
//    if (params.outputSelect == OutputSelect::HistogramStretch)
//        gpuFrame.download(outputFrame);

//    if (params.outputSelect == OutputSelect::Colorize)
//        gpuFrame.download(outputFrame);

    //sharpen(frame);
    gpuFrame.download(frame);
//	return;
}

void cudaDsp::equalizeHistogram(cv::cuda::GpuMat& frame)
{
    std::vector<cv::cuda::GpuMat> channels;
    cv::cuda::GpuMat img_hist_equalized;

    cv::cuda::cvtColor(frame, img_hist_equalized, CV_BGR2YCrCb); //change the color image from BGR to YCrCb format
    cv::cuda::split(img_hist_equalized,channels); //split the image into channels
    cv::cuda::equalizeHist(channels[0], channels[0]); //equalize histogram on the 1st channel (Y)
    cv::cuda::merge(channels,img_hist_equalized); //merge 3 channels including the modified 1st channel into one image
    cv::cuda::cvtColor(img_hist_equalized, frame, CV_YCrCb2BGR); //change the color image from YCrCb to BGR format (to display image properly)

}

void cudaDsp::frameAverage(cv::cuda::GpuMat& frame)
{
    static cv::cuda::GpuMat previousFrame;

    if (previousFrame.empty())
    {
        previousFrame = frame;
        return;
    }

    cv::cuda::GpuMat avg, diff;
    avg.create(frame.cols, frame.rows, CV_32FC3);
    diff.create(frame.cols, frame.rows, CV_32FC1);
    cv::cuda::addWeighted(frame, 0.5, previousFrame, 0.5, 0, avg);

    frame.copyTo(previousFrame);
}

void cudaDsp::edgeEnhance(cv::cuda::GpuMat& frame, TiDspParameters params)
{
    switch (params.edgeEnhanceType)
	{
		case EdgeEnhanceType::Sharpen:
		{
            cv::Mat kernel = (cv::Mat_<float>(3,3) <<
                     0,-1, 0,
                    -1, 5,-1,
                     0,-1, 0);

			cv::cuda::GpuMat alpha(frame.rows, frame.cols, CV_32FC1, double(0));
			cv::cuda::GpuMat frameFC3;
			frame.convertTo(frameFC3, CV_32FC3);

			std::vector<cv::cuda::GpuMat> channels;

			cv::cuda::split(frameFC3, channels); //split the image into channels
			channels.push_back(alpha);

			cv::cuda::GpuMat frameWithAlpha;
			cv::cuda::merge(channels, frameWithAlpha);

			cv::Ptr<cv::cuda::Filter> filter2D = cv::cuda::createLinearFilter(CV_32FC4, -1, kernel);
			filter2D->apply(frameWithAlpha, frameWithAlpha);

			cv::cuda::split(frameWithAlpha, channels); //split the image into channels
			channels.pop_back();

			cv::cuda::merge(channels, frameFC3);

			frameFC3.convertTo(frame, CV_8UC3);
			break;
		}

		case EdgeEnhanceType::Gaussian:
		{
			cv::Mat kernel = (cv::Mat_<float>(5,5) <<
							 -1,-1,-1,-1,-1,
							 -1, 2, 2, 2,-1,
							 -1, 2, 8, 2,-1,
							 -1, 2, 2, 2,-1,
							 -1,-1,-1,-1,-1) / 8.0;

			cv::cuda::GpuMat alpha(frame.rows, frame.cols, CV_32FC1, double(0));
			cv::cuda::GpuMat frameFC3;
			frame.convertTo(frameFC3, CV_32FC3);

			std::vector<cv::cuda::GpuMat> channels;

			cv::cuda::split(frameFC3, channels); //split the image into channels
			channels.push_back(alpha);

			cv::cuda::GpuMat frameWithAlpha;
			cv::cuda::merge(channels, frameWithAlpha);

			cv::Ptr<cv::cuda::Filter> filter2D = cv::cuda::createLinearFilter(CV_32FC4, -1, kernel);
			filter2D->apply(frameWithAlpha, frameWithAlpha);

			cv::cuda::split(frameWithAlpha, channels); //split the image into channels
			channels.pop_back();

			cv::cuda::merge(channels, frameFC3);

			frameFC3.convertTo(frame, CV_8UC3);

			break;
		}

		case EdgeEnhanceType::Unsharp_mask:
		{
            cv::Mat kernel = (cv::Mat_<float>(5,5) <<
                     1, 4,   6, 4, 1,
                     4,16,  24,16, 4,
                     6,24,-476,24, 6,
                     4,16,  24,16, 4,
                     1, 4,   6, 4, 1) / -256.0;

			cv::cuda::GpuMat alpha(frame.rows, frame.cols, CV_32FC1, double(0));
			cv::cuda::GpuMat frameFC3;
			frame.convertTo(frameFC3, CV_32FC3);

			std::vector<cv::cuda::GpuMat> channels;

			cv::cuda::split(frameFC3, channels); //split the image into channels
			channels.push_back(alpha);

			cv::cuda::GpuMat frameWithAlpha;
			cv::cuda::merge(channels, frameWithAlpha);

			cv::Ptr<cv::cuda::Filter> filter2D = cv::cuda::createLinearFilter(CV_32FC4, -1, kernel);
			filter2D->apply(frameWithAlpha, frameWithAlpha);

			cv::cuda::split(frameWithAlpha, channels); //split the image into channels
			channels.pop_back();

			cv::cuda::merge(channels, frameFC3);

			frameFC3.convertTo(frame, CV_8UC3);
            break;
		}

		default:
		{
			break;
		}
	}
    //frame.convertTo(frame, CV_8U);
    //cv::cuda::convertScaleAbs(frame, frame);
}



//__global__ void convolutionGPU(
//		std::vector<cv::cuda::GpuMat> inputChannels,
//		std::vector<cv::cuda::GpuMat> outputChannels,
//		cv::cuda::GpuMat kernel)
//{
//	int KERNEL_W = (2 * KERNEL_RADIUS + 1);
//    //////////////////////////////////////////////////////////////////////
//    // most slowest way to compute convolution
//    //////////////////////////////////////////////////////////////////////
//
//    // global mem address for this thread
//    const int gLoc = threadIdx.x +
//                    blockIdx.x * blockDim.x +
//                    threadIdx.y * dataW +
//                    blockIdx.y * blockDim.y * dataW;
//
//    float sum = 0;
//    float value = 0;
//
//    for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++) // row wise
//        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) // col wise
//        {
//            // check row first
//            if (blockIdx.x == 0 && (threadIdx.x + i) < 0) // left apron
//                value = 0;
//            else if ( blockIdx.x == (gridDim.x - 1) &&
//                        (threadIdx.x + i) > blockDim.x-1 ) // right apron
//                value = 0;
//            else
//            {
//                // check col next
//                if (blockIdx.y == 0 && (threadIdx.y + j) < 0) // top apron
//                    value = 0;
//                else if ( blockIdx.y == (gridDim.y - 1) &&
//                            (threadIdx.y + j) > blockDim.y-1 ) // bottom apron
//                    value = 0;
//                else // safe case
//                    value = d_Data[gLoc + i + j * dataW];
//            }
//            sum += value * d_Kernel[KERNEL_RADIUS + i] * d_Kernel[KERNEL_RADIUS + j];
//        }
//        d_Result[gLoc] = sum;
//}


void cudaDsp::eZoom(cv::cuda::GpuMat &frame, TiDspParameters params)
{
	/*
    cv::cuda::GpuMat dest;
    switch (params.ezoom)
    {
        case Ezoom::NoZoom:
            break;
        case Ezoom::x2:
            cv::cuda::pyrUp(frame, dest, cv::cuda::Size(frame.cols * 2, frame.rows * 2));
            moveFrame(frame, dest, params);
            break;
        case Ezoom::x4:
            cv::cuda::pyrUp(frame, dest, cv::cuda::Size(frame.cols * 2, frame.rows * 2));
            cv::cuda::pyrUp(dest, dest, cv::cuda::Size(dest.cols * 2, dest.rows * 2));
            moveFrame(frame, dest, params);
            break;
        case Ezoom::x8:
            cv::cuda::pyrUp(frame, dest, cv::cuda::Size(frame.cols * 2, frame.rows * 2));
            cv::cuda::pyrUp(dest, dest, cv::cuda::Size(dest.cols * 2, dest.rows * 2));
            cv::cuda::pyrUp(dest, dest, cv::cuda::Size(dest.cols * 2, dest.rows * 2));
            moveFrame(frame, dest, params);
            break;
    }
    */
}

void cudaDsp::moveFrame(const cv::cuda::GpuMat &frame, const cv::cuda::GpuMat &dest, TiDspParameters params)
{
	/*
    double availableXPixels, availableYPixels;
    double xMove, yMove;

    availableXPixels = dest.cols - frame.cols;
    availableYPixels = dest.rows - frame.rows;
    xMove = params.horisontalOffset / 100.0;
    yMove = params.verticalOffset / 100.0;
    dest(cv::cuda::Rect(xMove * availableXPixels, yMove * availableYPixels, frame.cols, frame.rows)).copyTo(frame);
    */
}
#endif

std::string cudaDsp::type2str(int type)
{
  std::string r;

  uchar depth = type & CV_MAT_DEPTH_MASK;
  uchar chans = 1 + (type >> CV_CN_SHIFT);

  switch ( depth ) {
    case CV_8U:  r = "8U"; break;
    case CV_8S:  r = "8S"; break;
    case CV_16U: r = "16U"; break;
    case CV_16S: r = "16S"; break;
    case CV_32S: r = "32S"; break;
    case CV_32F: r = "32F"; break;
    case CV_64F: r = "64F"; break;
    default:     r = "User"; break;
  }

  r += "C";
  r += (chans+'0');

  return r;
}
